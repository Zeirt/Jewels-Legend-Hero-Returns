#include <stdio.h>
#include "game_helper.h"

/*
	EN: Saves parameters of the current execution and the table in a .dat file. Returns true if success, false if fail.
	ES: Guarda parametros de la ejecuci�n actual y la tabla en un archivo .dat. Devuelve true si hay �xito, false si falla.
*/
bool saveData(int width, int height, bool isManual, int diffculty, char* table){
	FILE *saveData;

	saveData = fopen("save.dat", "W");
	//Open it as empty file for writing. That means we overwrite old save data.
	if (saveData == NULL) return false; //File couldn't open.

	fwrite(&width, sizeof(int), 1, saveData);
	fwrite(&height, sizeof(int), 1, saveData);
	fwrite(&isManual, sizeof(bool), 1, saveData);
	fwrite(&diffculty, sizeof(int), 1, saveData);
	fwrite(&table, sizeof(table), 1, saveData); //THIS MAY NEED TWEAKING

	//Finish writing and close the file
	fclose(saveData);
	return true;
}

/*
	EN: Loads the data saved by saveData() in order to put it back in the program to re-start from there. 
	If the parameters of the execution difer from save, it will abort the loading and give false. Success gives true.
	ES: Carga los datos guardados por saveData() para insertarlos de nuevo en el programa y reanudar desde ese estado.
	Si los parametros de la ejecucion son distintos al guardado, aborta y da false. Si hay exito devuelve true.
*/
bool loadData(int* width, int* height, bool* isManual, int* difficulty, char* table){
	FILE *saveData;

	saveData = fopen("save.dat", "r");
	//Open existing file for reading. If it doesn't exist, exit the function
	if (saveData == NULL) return false;

	int tWidth, tHeight, tDifficulty;
	bool tIsManual;

	//Check every parameter as you read. If it doesn't fit, return false
	fread(&tWidth, sizeof(int), 1, saveData);
	if (tWidth != *width) return false;
	fread(&tHeight, sizeof(int), 1, saveData);
	if (tHeight != *height) return false;
	fread(&tIsManual, sizeof(bool), 1, saveData);
	if (tIsManual != *isManual) return false;
	fread(&tDifficulty, sizeof(int), 1, saveData);
	if (tDifficulty != *difficulty) return false;
	//All fits if it reaches this point, write to the variables
	*width = tWidth;
	*height = tHeight;
	*isManual = tIsManual;
	*difficulty = tDifficulty;
	//I know we settled for passing this to the table directly, but truth to be told, it's unsafe. Returning string
	fread(&table, ((*width * *height) * sizeof(char)), 1, saveData);

	//Finish reading and close the file
	fclose(saveData);
	return true;
}